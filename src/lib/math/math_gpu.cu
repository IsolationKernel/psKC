#include "hip/hip_runtime.h"
#include <mass_ml/math/math.h>

#include <mass_ml/ds/ds_tmp_file.h>

#include <iostream>
#include <limits>

// CUDA Runtime
//#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>
//#include <helper_functions.h>

/*
 * The following CUDA kernel codes are from the CUDA samples with modification.
 *
 * - compute_dist_p_2 : 0_Simple/matrixMUL
 *
 */


constexpr uint16_t BLOCK_SIZE = 32;

namespace gpu_matrix_dense {

  __global__
  void compute_dist_p_2(double * C, double * A, double * B, int wA, int wB) {
  // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

  // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

  // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
    int bBegin = wA * BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
    double Csub = 0.0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
      __shared__ double As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
      __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
      As[ty][tx] = A[a + wA * ty + tx];
      Bs[tx][ty] = B[b + wA * ty + tx];

    // Synchronize to make sure the matrices are loaded
      __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll
      for (int k = 0; k < BLOCK_SIZE; ++k) {
        double val = As[ty][k] - Bs[k][tx];
        Csub += val * val;
//        Csub += std::pow(val, 2.0);
      }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
      __syncthreads();
    }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] += Csub;
  }

  __global__
  void do_sqrt(double * C, int wB) {
  // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

  // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = std::sqrt(C[c + wB * ty + tx]);
  }

  __global__
  void compute_NN(double * C, double * A, int wA, int wB) {
  }

}

namespace {

  struct stats_s {
    std::size_t cols;
    std::size_t a_rows;
    std::size_t b_rows;

    std::size_t act_cols;
    std::size_t act_a_rows;
    std::size_t act_b_rows;

    std::size_t blk_sz_cols;
    std::size_t blk_sz_a_rows;
    std::size_t blk_sz_b_rows;

    std::size_t blk_cnt_cols;
    std::size_t blk_cnt_a_rows;
    std::size_t blk_cnt_b_rows;
  };

  std::size_t max_cpu_mem_GiB{4ull * 1024 * 1024 * 1024};
  std::size_t max_gpu_mem_GiB{4ull * 1024 * 1024 * 1024};

  inline std::size_t compute_minimum_entries(std::size_t val) {
    return val + ((((val / BLOCK_SIZE) * BLOCK_SIZE) == val) ? 0 : BLOCK_SIZE - (val % BLOCK_SIZE));;
  }

  void compute_dist_stats(stats_s & stat) {
    stat.act_cols = compute_minimum_entries(stat.cols);
    stat.act_a_rows = compute_minimum_entries(stat.a_rows);
    stat.act_b_rows = compute_minimum_entries(stat.b_rows);

    stat.blk_sz_cols = BLOCK_SIZE;
    stat.blk_sz_a_rows = BLOCK_SIZE;
    stat.blk_sz_b_rows = BLOCK_SIZE;

    for (std::size_t i = 1; (i <= stat.act_cols) || (i <= stat.act_a_rows) || (i <= stat.act_b_rows); i++) {
      std::size_t blk_sz = i * BLOCK_SIZE;
      std::size_t temp_cols = (blk_sz < stat.act_cols) ? blk_sz : stat.act_cols;
      std::size_t temp_a_rows = (blk_sz < stat.act_a_rows) ? blk_sz : stat.act_a_rows;
      std::size_t temp_b_rows = (blk_sz < stat.act_b_rows) ? blk_sz : stat.act_b_rows;

      if ((8ull * ((temp_a_rows * temp_b_rows) + (temp_cols * temp_a_rows) + (temp_cols * temp_b_rows))) > max_gpu_mem_GiB) {
        break;
      }

      stat.blk_sz_cols = temp_cols;
      stat.blk_sz_a_rows = temp_a_rows;
      stat.blk_sz_b_rows = temp_b_rows;
    }

    stat.blk_cnt_cols = stat.act_cols / stat.blk_sz_cols;
    stat.blk_cnt_a_rows = stat.act_a_rows / stat.blk_sz_a_rows;
    stat.blk_cnt_b_rows = stat.act_b_rows / stat.blk_sz_b_rows;

    if ((stat.blk_cnt_cols * stat.blk_sz_cols) < stat.act_cols) {
      stat.blk_cnt_cols++;
    }

    if ((stat.blk_cnt_a_rows * stat.blk_sz_a_rows) < stat.act_a_rows) {
      stat.blk_cnt_a_rows++;
    }

    if ((stat.blk_cnt_b_rows * stat.blk_sz_b_rows) < stat.act_b_rows) {
      stat.blk_cnt_b_rows++;
    }
  }

  void load_data(double * dst, mass_ml::data_source_c const & src, std::size_t src_idx, std::size_t src_blk_sz, std::size_t col_idx, std::size_t col_blk_sz) {
    checkCudaErrors(hipMemset(dst, 0x00, sizeof(double) * src_blk_sz * col_blk_sz));

    std::vector<double> vals;
    vals.resize(src.cols(), 0.0);

    std::size_t src_offset = src_idx * src_blk_sz;
    std::size_t col_offset = col_idx * col_blk_sz;

    for (std::size_t i = 0; i < src_blk_sz; i++) {
      if ((src_offset + i) < src.rows()) {
        std::fill(vals.begin(), vals.end(), 0.0);
        src.load_row(vals, src_offset + i);

        for (std::size_t j = 0; j < col_blk_sz; j++) {
          if ((col_offset + j) < src.cols()) {
            dst[i * col_blk_sz + j] = vals[col_offset + j];
          }
        }
      }
    }
  }

  void load_data(double * dst, mass_ml::data_source_c const & src, std::size_t row_idx, std::size_t row_blk_sz, std::size_t col_idx, std::size_t col_blk_sz, std::size_t cols) {

  }

  void store_results(mass_ml::data_source_c & results, double * dist, std::size_t sz_a, std::size_t sz_b) {
    for (std::size_t i = 0; i < sz_a; i++) {
      std::vector<double> row;

      for (std::size_t j = 0; j < sz_b; j++) {
        row.push_back(dist[i * sz_b + j]);
      }

      results.store_row(row, "0");
    }
  }

  void store_results(mass_ml::data_source_c & dist, mass_ml::data_source_c const & results, std::size_t sz_a, std::size_t sz_b, std::size_t idx_a, std::size_t max_a, std::size_t max_b) {
    std::size_t blk_cnt = results.rows() / sz_a;

    std::size_t a_offset = idx_a * sz_a;

    std::vector<double> res_row;
    res_row.resize(sz_b, 0.0);

    for (std::size_t i = 0; i < sz_a; i++) {
      if ((a_offset + i) < max_a) {
        std::vector<double> row;

        for (std::size_t j = 0; j < blk_cnt; j++) {
          std::fill(res_row.begin(), res_row.end(), 0.0);
          results.load_row(res_row, j * sz_a + i);

          for (std::size_t k = 0; k < sz_b; k++) {
            if ((j * sz_b + k) < max_b) {
              row.push_back(res_row[k]);
            }
          }
        }

        dist.store_row(row, "");
      }
    }
  }

}

namespace mass_ml {


  void compute_distance(data_source_c & dist, data_source_c const & a, data_source_c const & b) {
    stats_s stat;
    stat.cols = a.cols();
    stat.a_rows = a.rows();
    stat.b_rows = b.rows();

    compute_dist_stats(stat);

    std::cout << "gpu::compute_distance:" << std::endl;
    std::cout << "- cols    : " << stat.cols << " (" << stat.act_cols << ") - " << stat.blk_sz_cols << " (" << stat.blk_cnt_cols << ")" << std::endl;;
    std::cout << "- rows (a): " << stat.a_rows << " (" << stat.act_a_rows << ") - " << stat.blk_sz_a_rows << " (" << stat.blk_cnt_a_rows << ")" << std::endl;;
    std::cout << "- rows (b): " << stat.b_rows << " (" << stat.act_b_rows << ") - " << stat.blk_sz_b_rows << " (" << stat.blk_cnt_b_rows << ")" << std::endl;;

    double * temp_dists;
    double * temp_a_rows;
    double * temp_b_rows;
    checkCudaErrors(hipMallocManaged(&temp_dists, sizeof(double) * stat.blk_sz_a_rows * stat.blk_sz_b_rows));
    checkCudaErrors(hipMallocManaged(&temp_a_rows, sizeof(double) * stat.blk_sz_a_rows * stat.blk_sz_cols));
    checkCudaErrors(hipMallocManaged(&temp_b_rows, sizeof(double) * stat.blk_sz_b_rows * stat.blk_sz_cols));

    for (std::size_t i = 0; i < stat.blk_cnt_a_rows; i++) {
      std::unique_ptr<data_source_c> results = data_source_c::make_unique<ds_tmp_file_c>(stat.blk_sz_b_rows);

      for (std::size_t j = 0; j < stat.blk_cnt_b_rows; j++) {
        checkCudaErrors(hipMemset(temp_dists, 0x00, sizeof(double) * stat.blk_sz_a_rows * stat.blk_sz_b_rows));

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(stat.blk_sz_b_rows / block.x, stat.blk_sz_a_rows / block.y);

        for (std::size_t k = 0; k < stat.blk_cnt_cols; k++) {
          load_data(temp_a_rows, a, i, stat.blk_sz_a_rows, k, stat.blk_sz_cols);
          load_data(temp_b_rows, b, j, stat.blk_sz_b_rows, k, stat.blk_sz_cols);

          gpu_matrix_dense::compute_dist_p_2<<<grid, block>>>(temp_dists, temp_a_rows, temp_b_rows, stat.blk_sz_cols, stat.blk_sz_b_rows);
          getLastCudaError("Kernel execution failed: gpu_matrix_dense::compute_distance");
          hipDeviceSynchronize();
        }

        gpu_matrix_dense::do_sqrt<<<grid, block>>>(temp_dists, stat.blk_sz_b_rows);
        getLastCudaError("Kernel execution failed: gpu_matrix_dense::do_sqrt");
        hipDeviceSynchronize();

        store_results(*results, temp_dists, stat.blk_sz_a_rows, stat.blk_sz_b_rows);
      }

      store_results(dist, *results, stat.blk_sz_a_rows, stat.blk_sz_b_rows, i, a.rows(), b.rows());
    }

    hipFree(temp_a_rows);
    hipFree(temp_b_rows);
    hipFree(temp_dists);
  }

  void init_math(std::size_t cpu_mem_GiB, std::size_t gpu_mem_GiB) {
    max_cpu_mem_GiB = cpu_mem_GiB;
    max_gpu_mem_GiB = gpu_mem_GiB;
  }

  void find_nearest_neighbour(data_source_c const & dist, data_source_c & idx, std::size_t blk_sz, bool inc_zero) {
    std::size_t entries = dist.cols() / blk_sz;

    std::vector<double> x;
    x.resize(dist.cols(), 0.0);

    for (std::size_t i = 0; i < dist.rows(); i++) {
      std::fill(x.begin(), x.end(), 0.0);
      dist.load_row(x, i);

      std::vector<double> idx_row;

      for (std::size_t j = 0; j < entries; j++) {
        std::size_t offset = blk_sz * j;

        double best_dist = std::numeric_limits<double>::max();
        std::size_t best_idx = 0;

        for (std::size_t k = 0; k < blk_sz; k++) {
  #ifdef NDEBUG
          double val = x[offset + k];
  #else
          double val = x.at(offset + k);
  #endif

          if (!inc_zero && (val == 0.0)) {
            continue;
          }

          if (val < best_dist) {
            best_dist = val;
            best_idx = k;
          }
        }

        idx_row.push_back(best_idx);
      }

      idx.store_row(idx_row, "0");
    }
  }

}
